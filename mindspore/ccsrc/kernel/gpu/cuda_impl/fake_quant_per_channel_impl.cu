#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>
#include "fake_quant_per_channel_impl.cuh"
#include "device/gpu/cuda_common.h"

/**
 * Find the nudge min, max and scale value as output.
 * @param input_min array
 * @param input_max array
 * @param quant_min 1 << bit -1
 * @param quant_max 0
 * @param nudge_min array
 * @param nudge_max array
 * @param scale array
 * @param channel_num
 * @return
 */
__global__ void NudgeMinMaxPerChannel(const float *input_min, const float *input_max, const float quant_min,
                                      const float quant_max, float *nudge_min, float *nudge_max, float *scale,
                                      int channel_num) {
  float zp_from_min = 0.f;
  float nudge_zp = 0.f;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < channel_num; i += blockDim.x * gridDim.x) {
    if ((quant_max - quant_min) == 0 || (input_max[i] - input_min[i]) == 0) {
      scale[i] = 0.f;
      zp_from_min = 0.f;
    } else {
      scale[i] = (input_max[i] - input_min[i]) / (quant_max - quant_min);
      zp_from_min = quant_min - input_min[i] / scale[i];
    }

    if (zp_from_min <= quant_min) {
      nudge_zp = quant_min;
    } else if (zp_from_min >= quant_max) {
      nudge_zp = quant_max;
    } else {
      nudge_zp = round(zp_from_min);
    }

    nudge_min[i] = (quant_min - nudge_zp) * (scale[i]);
    nudge_max[i] = (quant_max - nudge_zp) * (scale[i]);
  }
}

void CalNudgePerChannel(const float *input_min, const float *input_max, const float quant_min, const float quant_max,
                        float *nudge_min, float *nudge_max, float *scale, const int channel_num,
                        hipStream_t hip_stream) {
  NudgeMinMaxPerChannel<<<GET_BLOCKS(channel_num), GET_THREADS, 0, hip_stream>>>(
    input_min, input_max, quant_min, quant_max, nudge_min, nudge_max, scale, channel_num);
}

/**
 * Calulate fake quant output accroding by nudge min, nudge max, nudge scale.
 * @param input - array
 * @param output - array
 * @param total_size - int, purpose for cal the per chanel number in filters
 * @param channel_size - int, purpose for cal the per channel number in filters
 * @param nudge_min - array
 * @param nudge_max - array
 * @param scale - array
 * @return
 */
__global__ void FakeQuantizePerChannel(const float *input, float *output, const int total_size, const int channel_size,
                                       const float *nudge_min, const float *nudge_max, const float *scale,
                                       bool symmetric) {
  float input_x = 0.f;
  int nudge_input = 0;
  int channel_idx = 0;
  int per_channel_num = total_size / channel_size;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
    input_x = input[i];
    channel_idx = floor(static_cast<double>(i) / static_cast<double>(per_channel_num));
    // clamp input x
    if (input_x < nudge_min[channel_idx]) {
      input_x = nudge_min[channel_idx];
    }
    if (input_x > nudge_max[channel_idx]) {
      input_x = nudge_max[channel_idx];
    }
    // clamp shift
    nudge_input = floor((input_x - nudge_min[channel_idx]) / scale[channel_idx] + 0.5f);

    // quantize
    output[i] = nudge_input * scale[channel_idx] + nudge_min[channel_idx];
  }
}

void CalFakeQuantizePerChannel(const float *input, float *output, const int total_size, const int channel_size,
                               const float *nudge_min, const float *nudge_max, const float *scale, bool symmetric,
                               hipStream_t hip_stream) {
  FakeQuantizePerChannel<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(
    input, output, total_size, channel_size, nudge_min, nudge_max, scale, symmetric);
}

/**
 * UpdateInputMinMaxPerChannel or UpdateInputMinMaxPerChannel With EMA.
 * @param input_min
 * @param input_max
 * @param min
 * @param max
 * @return
 */
__global__ void UpdateInputMinMaxPerChannel(float *input_min, float *input_max, float *input, int channels,
                                            int per_channel_nums, bool ema, float ema_decay) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < channels; i += blockDim.x * gridDim.x) {
    thrust::pair<float *, float *> sum =
      thrust::minmax_element(thrust::device, input + i * per_channel_nums, input + per_channel_nums * (i + 1));
    if (ema) {
      input_min[i] = ema_decay * sum.first[0] + (1 - ema_decay) * input_min[i];
      input_max[i] = ema_decay * sum.second[0] + (1 - ema_decay) * input_max[i];
    } else {
      input_min[i] = sum.first[0];
      input_max[i] = sum.second[0];
    }
    input_min[i] = input_min[i] > 0 ? 0 : input_min[i];
    input_max[i] = input_max[i] < 0 ? 0 : input_max[i];
  }
}

__global__ void UpdateInputMinMaxPerChannelWithEMA(float *input_min, float *input_max, float min, float max,
                                                   const float decay) {
  *input_min = decay * (min) + (1 - decay) * (*input_min);
  *input_max = decay * (max) + (1 - decay) * (*input_max);
}

void CalMinMaxPerChannel(float *input, float *input_min, float *input_max, const int total_size, const int channel_size,
                         const float ema_decay, const bool ema, hipStream_t hip_stream) {
  int per_channel_num = total_size / channel_size;
  UpdateInputMinMaxPerChannel<<<GET_BLOCKS(channel_size), GET_THREADS, 0, hip_stream>>>(
    input_min, input_max, input, channel_size, per_channel_num, ema, ema_decay);
}

__global__ void FakeQuantizePerChannelGrad(const float *input, const float *gradient, float *output,
                                           const int total_size, const int channel_size, const float *nudge_min,
                                           const float *nudge_max) {
  int channel_idx = 0;
  int per_channel_num = total_size / channel_size;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < total_size; i += blockDim.x * gridDim.x) {
    channel_idx = floor(static_cast<double>(i) / static_cast<double>(per_channel_num));
    if (input[i] < nudge_min[channel_idx] || input[i] > nudge_max[channel_idx]) {
      output[i] = 0;
    } else {
      output[i] = gradient[i];
    }
  }
}

void CalFakeQuantizePerChannelGrad(const float *input, const float *gradient, float *output, const int total_num,
                                   const int channel_num, const float *nudge_min, const float *nudge_max,
                                   hipStream_t hip_stream) {
  FakeQuantizePerChannelGrad<<<GET_BLOCKS(channel_num), GET_THREADS, 0, hip_stream>>>(
    input, gradient, output, total_num, channel_num, nudge_min, nudge_max);
}
